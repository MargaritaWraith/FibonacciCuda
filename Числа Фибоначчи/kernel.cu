#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Math.h>
#include <stdio.h>

hipError_t PhibWithCuda(int *Phib, unsigned int size);


__global__ void PhibKernel(int *Phib)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Phib[i] = int(pow(1.6180339887, (int)i) / 2.236067977 + 0.5); // pow(1.6180339887,(int)i)
}

int main()
{
	const int arraySize = 100;
	int Phib[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = PhibWithCuda(Phib, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (int i = 0; i < arraySize; i++)
	{
		printf("Phib[%d] = %d\n", i + 1, Phib[i]);
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t PhibWithCuda(int *Phib, unsigned int size)
{
	int *dev_Phib = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_Phib, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	dim3 block(32, 1);
	dim3 grid((size / 32), 1);
	PhibKernel << <grid, block >> > (dev_Phib);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Phib, dev_Phib, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_Phib);

	return cudaStatus;
}
