#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <Math.h>
#include <stdio.h>
#include <locale>

hipError_t PhibWithCuda(unsigned long long *Phib, unsigned int size);


__global__ void PhibKernel(unsigned long long *Phib)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Phib[i] = unsigned long long(pow(1.6180339887498948482045868343656, i) / 2.2360679774997896964091736687313 + 0.5);

	// 1/sqrt(5) = 0.44721359549995793928183473374626
	// Phi = (1+sqrt(5))/2 = 1.6180339887498948482045868343656  ������� �������!  +/-Phi - �������� ������� ������������������� ��������� x^{2}-x-1=0
	// sqrt(5) = 2.2360679774997896964091736687313
}

int main()
{
	setlocale(LC_ALL, "Russian");
	const int arraySize = 64;
	unsigned long long Phib[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = PhibWithCuda(Phib, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (int i = 0; i < arraySize; i++)
	{
		printf("Phib[%d] = %llu\n", i + 1, Phib[i]);
	}


	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t PhibWithCuda(unsigned long long *Phib, unsigned int size)
{
	unsigned long long *dev_Phib = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers.
	cudaStatus = hipMalloc((void**)&dev_Phib, size * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	PhibKernel << <2, 32 >> > (dev_Phib);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(Phib, dev_Phib, size * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_Phib);

	return cudaStatus;
}
